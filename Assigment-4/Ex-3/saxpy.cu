
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<unistd.h>
#include<sys/time.h>
#include<math.h>
#include<iostream>

#ifndef NUM_THREADS_PER_BLOCK
#define NUM_THREADS_PER_BLOCK 256
#endif

typedef struct timeval tval;
double get_elapsed(tval t0, tval t1);

void saxpy(int n, const float a, float *x, float *y)
{
    for (int i=0; i<n; i++)
    {
        y[i] = a*x[i] + y[i];
    }
}

__global__
void saxpy_cuda(int n,const float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char *argv[])
{   
    int           N                     = 10000;
    const float   a                     = 2.0;
    tval          t[2]                  = { 0 };
    double        elapsed[2]            = { 0 };
    float         *x, *y, *d_x, *d_y;

    int c;
    while ((c = getopt(argc, argv, "n:p:h")) != -1)
        switch (c)
        {
            case 'n':
                N = atoi(optarg);
                break;
            // case 'p':
            //     n_threads = atoi(optarg);
            //     break;
            case 'h':
                printf(
                    "Options:\n-n SIZE\t\tNum Particle\n-p NTHREAD\tNumber of threads\n");
                exit(1);
            case '?':
                break;
        }


    int num_blocks = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    printf("Num_blocks: %2d  Num_threads_per_block: %2d \n", num_blocks, NUM_THREADS_PER_BLOCK);

    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    for (int i = 0; i < N; i++) 
    {
       x[i] = ((float)rand() / (float)RAND_MAX)*2 -1;
       y[i] = ((float)rand() / (float)RAND_MAX)*2 -1;
    }

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Launch the CPU version
    printf("Computing SAXPY on the CPU...");
    gettimeofday(&t[0], NULL);
    saxpy(N, a, x, y);
    gettimeofday(&t[1], NULL);        
    elapsed[0] = get_elapsed(t[0], t[1]);
    printf("Done!\n");

    // Launch the GPU version
    printf("Computing SAXPY on the GPU...");
    gettimeofday(&t[0], NULL);
    saxpy_cuda<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(N, a, d_x, d_y);
    // Copy array y back to HOST and store in array x
    hipMemcpy(x, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    gettimeofday(&t[1], NULL);    
    elapsed[1] = get_elapsed(t[0], t[1]);
    printf("Done!\n");

    printf("Comparing the output for each implementation...");
    float error = 0.0f;
    for (int i = 0; i < N; i++)
        error = fmax(error, fabs(y[i]-x[i]));

    if (error < 1e-8){
        printf("Correct!\n");
    }
    else{
        printf("Not correct!\n");
    }
    printf("Elapsed CPU (ms): %f / Elapsed GPU (ms): %f\n", elapsed[0], elapsed[1]);

    // Free memory
    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}

double get_elapsed(tval t0, tval t1)
{
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L + (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}