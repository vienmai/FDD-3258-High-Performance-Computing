
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<unistd.h>
#include<sys/time.h>
#include<math.h>
#include<iostream>

struct Particle{
    float px;
    float py;
    float pz;
    float vx;
    float vy;
    float vz;
};

typedef struct timeval tval;
double get_elapsed(tval t0, tval t1);

void get_input_data(struct Particle **particles, int N, const int seed);
void kernel(struct Particle *particles, int N, int n_steps, const double delta_t, const int seed);

__global__
void kernel_cuda(struct Particle *particles, int N, int n_steps, const double delta_t, const int seed);
__host__ __device__ 
float rand_float(const int seed, int particle, int iter, int N);


#define HLINE "-------------------------------------------------------------\n"

int main(int argc, char *argv[])
{   
    int c;
    int N=1000;
    int n_threads=256; 
    int n_steps=100;
    const int seed=2020;
    const double delta_t=0.05;
    tval t[2] = {0};
    double elapsed[2] = {0};

    while ((c = getopt(argc, argv, "n:p:s:h")) != -1)
        switch (c)
        {
            case 'n':
                N = atoi(optarg);
                break;
            case 'p':
                n_threads = atoi(optarg);
                break;
            case 's':
                n_steps = atoi(optarg);
                break;
            case 'h':
                printf(
                    "Options:\n-n SIZE\t\tNum Particle\n-s ITERS\tNum Iters\n-p NTHREAD\tNumber of threads\n");
                exit(1);
            case '?':
                break;
        }

    struct Particle* particles;
    struct Particle* d_particles;
    get_input_data(&particles, N, seed);
    
    hipMalloc(&d_particles, N*sizeof(struct Particle));
    
    // For storing the final result copied from Device to Host
    struct Particle* particles_=(struct Particle*)malloc(N*sizeof(struct Particle));
    hipMemcpy(d_particles, particles, N*sizeof(struct Particle), hipMemcpyHostToDevice);

    int n_blocks = (N + n_threads - 1) / n_threads;

    printf(HLINE);
    printf("  N\tNum steps\tNum Threads\tNum Blocks\n");
    printf("%4d %8d %14d %16d\n", N, n_steps, n_threads, n_blocks);
    printf(HLINE);
    
    // Launch the CPU version
    printf("Running particle simulation on the CPU...");
    gettimeofday(&t[0], NULL);
    kernel(particles, N, n_steps, delta_t, seed);
    gettimeofday(&t[1], NULL);
    elapsed[0] = get_elapsed(t[0], t[1]);
    printf("Done!\n");

    // Launch the GPU version
    printf("Running particle simulation on the GPU...");
    gettimeofday(&t[0], NULL);
    for (int iter = 1; iter <= n_steps; iter++)
        kernel_cuda<<<n_blocks, n_threads>>>(d_particles, N, iter, delta_t, seed);

    hipMemcpy(particles_, d_particles, N*sizeof(struct Particle), hipMemcpyDeviceToHost);
    gettimeofday(&t[1], NULL);    
    elapsed[1] = get_elapsed(t[0], t[1]);
    printf("Done!\n");

    printf("Comparing the output for each implementation...");
    float error = 0.0f;
    for (int i = 0; i < N; i++)
        error = fmax(error, fabs(particles[i].px - particles_[i].px));

    if (error < 1e-8){
        printf("Correct!\n");
    }
    else{
        printf("Not correct!\n");
    }
    printf("Elapsed CPU (ms): %f / Elapsed GPU (ms): %f\n", elapsed[0], elapsed[1]);
    printf(HLINE);

    free(particles);
    free(particles_);
    hipFree(d_particles);
    return 0;
}

__host__ __device__
float rand_float(const int seed, int particle, int iter, int N)
{
  float result = (seed * particle + iter) % N;
  return result;
}

__global__
void kernel_cuda(struct Particle *particles, int N, int iter, const double delta_t, const int seed)
{
    int q = blockIdx.x * blockDim.x + threadIdx.x;

    if(q < N){
        // particles[q].vx = (rand() / (double)(RAND_MAX)) * 2 - 1;
        // particles[q].vy = (rand() / (double)(RAND_MAX)) * 2 - 1;
        // particles[q].vz = (rand() / (double)(RAND_MAX)) * 2 - 1;
        particles[q].vx = rand_float(seed, q, iter, N) + 0.0f;
        particles[q].vy = rand_float(seed, q, iter, N) + 0.1f;
        particles[q].vz = rand_float(seed, q, iter, N) + 0.2f;

        particles[q].px += delta_t * particles[q].vx;
        particles[q].py += delta_t * particles[q].vy;
        particles[q].pz += delta_t * particles[q].vz;
    }
}


void kernel(struct Particle *particles, int N, int n_steps, const double delta_t, const int seed)
{
    for (int iter = 1; iter <= n_steps; iter++)
    {
        for (int q = 0; q < N; q++)
        {
            // particles[q].vx = (rand() / (double)(RAND_MAX)) * 2 - 1;
            // particles[q].vy = (rand() / (double)(RAND_MAX)) * 2 - 1;
            // particles[q].vz = (rand() / (double)(RAND_MAX)) * 2 - 1;
            particles[q].vx = rand_float(seed, q, iter, N) + 0.0f;
            particles[q].vy = rand_float(seed, q, iter, N) + 0.1f;
            particles[q].vz = rand_float(seed, q, iter, N) + 0.2f;

            particles[q].px += delta_t * particles[q].vx;
            particles[q].py += delta_t * particles[q].vy;
            particles[q].pz += delta_t * particles[q].vz;
        }
    }
}

void get_input_data(struct Particle **particles, int N, const int seed )
{
    struct Particle* particle_t = (struct Particle*)malloc(N*sizeof(struct Particle));
    srand(seed); 

    for(int q=0; q < N; q++)
    {
        particle_t[q].px = (rand() / (double)(RAND_MAX)) * 2 - 1;
        particle_t[q].py = (rand() / (double)(RAND_MAX)) * 2 - 1;
        particle_t[q].pz = (rand() / (double)(RAND_MAX)) * 2 - 1;

        particle_t[q].vx = (rand() / (double)(RAND_MAX)) * 2 - 1;
        particle_t[q].vy = (rand() / (double)(RAND_MAX)) * 2 - 1;
        particle_t[q].vz = (rand() / (double)(RAND_MAX)) * 2 - 1;
    }
    *particles = particle_t;
}

double get_elapsed(tval t0, tval t1)
{
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L + (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}