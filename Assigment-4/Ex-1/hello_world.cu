
#include <hip/hip_runtime.h>
#include<stdio.h>

#define NUM_THREADS_PER_BLOCK 256

__global__
void print_hello()
{   
    int idx = threadIdx.x;
    printf("Hello World! My threadId is %d\n", idx);
}

int main()
{
    print_hello<<<1, NUM_THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
    return 0;
}